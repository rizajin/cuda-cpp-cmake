#include "hip/hip_runtime.h"
#include "kernel.h"
#include <stdexcept>

#include <hip/hip_runtime.h>
#include <>

__global__ void addKernel(const int* a, const int* b, int* result)
{
    int i = threadIdx.x;
    result[i] = a[i] + b[i];
}

void DeAlloc(int* a, int* b, int* c)
{
    hipError_t e{ hipError_t::hipSuccess };

    e = hipFree(a);
    if (e != hipError_t::hipSuccess) {
        //msg

    }
    e = hipFree(b);
    if (e != hipError_t::hipSuccess) {
        //msg

    }
    e = hipFree(c);
    if (e != hipError_t::hipSuccess) {
        //msg

    }
}

hipError_t addWithCuda(const std::vector<int>& aset, const std::vector<int>& bset, std::vector<int>& result)
{
    int* a = 0;
    int* b = 0;
    int* res = 0;
    std::vector<int*> cudaObjects;

    hipError_t cudaStatus{ hipError_t::hipSuccess };

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipError_t::hipSuccess)
    {
        throw std::runtime_error("hipSetDevice Failed");
    }

    cudaStatus = hipMalloc((void**)&a, sizeof(int) * aset.size());
    if (cudaStatus != hipError_t::hipSuccess)
    {
        DeAlloc(a, b, res);
        throw std::runtime_error("hipMalloc for a failed");
    }
    cudaStatus = hipMalloc((void**)&b, sizeof(int) * bset.size());
    if (cudaStatus != hipError_t::hipSuccess)
    {
        DeAlloc(a, b, res);
        throw std::runtime_error("hipMalloc for b failed");
    }
    cudaStatus = hipMalloc((void**)&res, sizeof(int) * result.size());
    if (cudaStatus != hipError_t::hipSuccess)
    {
        DeAlloc(a, b, res);
        throw std::runtime_error("hipMalloc for res failed");
    }

    cudaStatus = hipMemcpy(a, aset.data(), sizeof(int) * aset.size(), hipMemcpyHostToDevice);
    if (cudaStatus != hipError_t::hipSuccess)
    {
        DeAlloc(a, b, res);
        throw std::runtime_error("hipMemcpy for a failed");
    }
    cudaStatus = hipMemcpy(b, bset.data(), sizeof(int) * bset.size(), hipMemcpyHostToDevice);
    if (cudaStatus != hipError_t::hipSuccess)
    {
        DeAlloc(a, b, res);
        throw std::runtime_error("hipMemcpy for a failed");
    }

    addKernel<<<1, aset.size()>>>(a, b, res);
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipError_t::hipSuccess)
    {
        DeAlloc(a, b, res);
        std::string e = "addKernel launch failed: ";
        e.append(hipGetErrorString(cudaStatus));
        throw std::runtime_error(e);
    }

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipError_t::hipSuccess)
    {
        DeAlloc(a, b, res);
        std::string e = "hipDeviceSynchronize error after addKernel launch: ";
        e.append(hipGetErrorString(cudaStatus));
        throw std::runtime_error(e);
    }

    cudaStatus = hipMemcpy(result.data(), res, sizeof(int) * aset.size(), hipMemcpyDeviceToHost);
    if (cudaStatus != hipError_t::hipSuccess)
    {
        DeAlloc(a, b, res);
        throw std::runtime_error("hipMemcpy failed for res");
    }

    return cudaStatus;
}

void ExecuteCuda(const std::vector<int>& setA, const std::vector<int>& setB, std::vector<int>& result)
{
    result.resize(setA.size());
    hipError_t cudaStatus = addWithCuda(setA, setB, result);
    if (cudaStatus != hipError_t::hipSuccess)
    {
        throw std::runtime_error("addWithCuda failed");
    }

    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipError_t::hipSuccess)
    {
        throw std::runtime_error("hipDeviceReset failed");
    }
}